/**
 * cuSPARSE CSR SpMM implementation for the ReorderingSurvey2025 framework
 * Performs sparse matrix-matrix multiplication A * B = C where A is sparse CSR
 */

#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <string>
#include <cmath>
#include <stdexcept>
#include <sstream>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

// Error checking macros
#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(err) << std::endl; \
            exit(1); \
        } \
    } while(0)

#define CHECK_CUSPARSE(call) \
    do { \
        hipsparseStatus_t err = call; \
        if (err != HIPSPARSE_STATUS_SUCCESS) { \
            std::cerr << "cuSPARSE error at " << __FILE__ << ":" << __LINE__ << " - " << err << std::endl; \
            exit(1); \
        } \
    } while(0)

struct MatrixMarketHeader {
    int rows, cols, nnz;
    bool is_symmetric;
};

// Parse Matrix Market header
MatrixMarketHeader parseMatrixMarketHeader(const std::string& filename) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        throw std::runtime_error("Cannot open matrix file: " + filename);
    }

    std::string line;
    MatrixMarketHeader header;
    header.is_symmetric = false;

    // Read header line
    std::getline(file, line);
    if (line.find("%%MatrixMarket") != 0) {
        throw std::runtime_error("Invalid Matrix Market format");
    }
    if (line.find("symmetric") != std::string::npos) {
        header.is_symmetric = true;
    }

    // Skip comments
    while (std::getline(file, line) && line[0] == '%');

    // Read dimensions
    std::istringstream iss(line);
    iss >> header.rows >> header.cols >> header.nnz;

    return header;
}

// Load CSR matrix from Matrix Market format
void loadMatrixMarket(const std::string& filename, 
                     std::vector<int>& rowPtr, 
                     std::vector<int>& colIdx, 
                     std::vector<double>& values) {
    
    MatrixMarketHeader header = parseMatrixMarketHeader(filename);
    
    std::ifstream file(filename);
    std::string line;
    
    // Skip to data
    while (std::getline(file, line) && line[0] == '%');
    std::getline(file, line); // Skip dimension line
    
    // Read coordinate format
    std::vector<std::vector<std::pair<int, double>>> rows(header.rows);
    
    int row, col;
    double val;
    for (int i = 0; i < header.nnz; ++i) {
        file >> row >> col >> val;
        rows[row-1].push_back({col-1, val}); // Convert to 0-based indexing
        
        // Handle symmetric matrices
        if (header.is_symmetric && row != col) {
            rows[col-1].push_back({row-1, val});
        }
    }
    
    // Convert to CSR format
    rowPtr.resize(header.rows + 1);
    rowPtr[0] = 0;
    
    for (int i = 0; i < header.rows; ++i) {
        // Sort columns in each row
        std::sort(rows[i].begin(), rows[i].end());
        
        for (const auto& entry : rows[i]) {
            colIdx.push_back(entry.first);
            values.push_back(entry.second);
        }
        rowPtr[i + 1] = colIdx.size();
    }
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <matrix.mtx> <output_file> [alpha] [beta] [num_cols_B]" << std::endl;
        return 1;
    }

    std::string matrix_file = argv[1];
    std::string output_file = argv[2];
    double alpha = (argc > 3) ? std::stod(argv[3]) : 1.0;
    double beta = (argc > 4) ? std::stod(argv[4]) : 0.0;
    int num_cols_B = (argc > 5) ? std::stoi(argv[5]) : 64; // Default dense matrix B with 64 columns

    try {
        // Load matrix A from file
        std::vector<int> h_rowPtr, h_colIdx;
        std::vector<double> h_values;
        loadMatrixMarket(matrix_file, h_rowPtr, h_colIdx, h_values);

        int num_rows_A = h_rowPtr.size() - 1;
        int num_cols_A = *std::max_element(h_colIdx.begin(), h_colIdx.end()) + 1;
        int nnz_A = h_values.size();

        std::cout << "Matrix A: " << num_rows_A << "x" << num_cols_A << " with " << nnz_A << " non-zeros" << std::endl;

        // Initialize cuSPARSE
        hipsparseHandle_t handle;
        CHECK_CUSPARSE(hipsparseCreate(&handle));

        // Allocate device memory for matrix A (CSR)
        int *d_rowPtr, *d_colIdx;
        double *d_values;
        CHECK_CUDA(hipMalloc(&d_rowPtr, (num_rows_A + 1) * sizeof(int)));
        CHECK_CUDA(hipMalloc(&d_colIdx, nnz_A * sizeof(int)));
        CHECK_CUDA(hipMalloc(&d_values, nnz_A * sizeof(double)));

        CHECK_CUDA(hipMemcpy(d_rowPtr, h_rowPtr.data(), (num_rows_A + 1) * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_colIdx, h_colIdx.data(), nnz_A * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_values, h_values.data(), nnz_A * sizeof(double), hipMemcpyHostToDevice));

        // Create sparse matrix A
        hipsparseSpMatDescr_t matA;
        CHECK_CUSPARSE(hipsparseCreateCsr(&matA, num_rows_A, num_cols_A, nnz_A,
                                        d_rowPtr, d_colIdx, d_values,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

        // Create dense matrix B (random values)
        double *d_B, *d_C;
        CHECK_CUDA(hipMalloc(&d_B, num_cols_A * num_cols_B * sizeof(double)));
        CHECK_CUDA(hipMalloc(&d_C, num_rows_A * num_cols_B * sizeof(double)));

        // Initialize B with random values on GPU - using simple constant for reproducibility
        std::vector<double> h_B(num_cols_A * num_cols_B, 1.0);
        CHECK_CUDA(hipMemcpy(d_B, h_B.data(), num_cols_A * num_cols_B * sizeof(double), hipMemcpyHostToDevice));

        // Create dense matrices B and C
        hipsparseDnMatDescr_t matB, matC;
        CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, num_cols_A, num_cols_B, num_cols_A, d_B, HIP_R_64F, HIPSPARSE_ORDER_COL));
        CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, num_rows_A, num_cols_B, num_rows_A, d_C, HIP_R_64F, HIPSPARSE_ORDER_COL));

        // Allocate workspace
        size_t bufferSize;
        CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                              &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize));

        void *d_buffer;
        CHECK_CUDA(hipMalloc(&d_buffer, bufferSize));

        // Warm-up run
        CHECK_CUSPARSE(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                   &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, d_buffer));
        CHECK_CUDA(hipDeviceSynchronize());

        // Timing runs
        const int num_trials = 10;
        auto start = std::chrono::high_resolution_clock::now();

        for (int trial = 0; trial < num_trials; ++trial) {
            CHECK_CUSPARSE(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, d_buffer));
        }
        CHECK_CUDA(hipDeviceSynchronize());

        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
        double avg_time_ms = duration.count() / (1000.0 * num_trials);

        // Calculate GFLOPS (2 * nnz * num_cols_B for SpMM)
        double gflops = (2.0 * nnz_A * num_cols_B) / (avg_time_ms * 1e6);

        // Write results
        std::ofstream outfile(output_file);
        outfile << "avg_time_ms," << avg_time_ms << std::endl;
        outfile << "gflops," << gflops << std::endl;
        outfile << "nnz," << nnz_A << std::endl;
        outfile << "num_rows," << num_rows_A << std::endl;
        outfile << "num_cols," << num_cols_A << std::endl;
        outfile << "num_cols_B," << num_cols_B << std::endl;
        outfile.close();

        std::cout << "Average time: " << avg_time_ms << " ms" << std::endl;
        std::cout << "Performance: " << gflops << " GFLOPS" << std::endl;

        // Cleanup
        hipsparseDestroySpMat(matA);
        hipsparseDestroyDnMat(matB);
        hipsparseDestroyDnMat(matC);
        hipsparseDestroy(handle);

        hipFree(d_rowPtr);
        hipFree(d_colIdx);
        hipFree(d_values);
        hipFree(d_B);
        hipFree(d_C);
        hipFree(d_buffer);

    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}